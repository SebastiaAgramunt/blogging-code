#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    std::srand(std::time(0));
    std::ofstream csv("timings.csv");
    csv << "N,LoadingTime(ms),CalculationTime(ms)\n";

    int sizes[] = {1024, 4096, 16384, 65536, 262144, 1048576}; // Example sizes

    for (int s = 0; s < sizeof(sizes)/sizeof(sizes[0]); ++s) {
        int N = sizes[s];
        std::vector<float> h_a(N), h_b(N), h_c(N);

        for (int i = 0; i < N; ++i) {
            h_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
            h_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
        }

        float *d_a, *d_b, *d_c;
        hipMalloc(&d_a, N * sizeof(float));
        hipMalloc(&d_b, N * sizeof(float));
        hipMalloc(&d_c, N * sizeof(float));

        hipEvent_t startLoad, stopLoad, startCalc, stopCalc;
        hipEventCreate(&startLoad);
        hipEventCreate(&stopLoad);
        hipEventCreate(&startCalc);
        hipEventCreate(&stopCalc);

        // Measure loading time (host to device + device to host)
        hipEventRecord(startLoad);
        hipMemcpy(d_a, h_a.data(), N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b.data(), N * sizeof(float), hipMemcpyHostToDevice);
        hipEventRecord(stopLoad);
        hipEventSynchronize(stopLoad);

        float loadTime = 0;
        hipEventElapsedTime(&loadTime, startLoad, stopLoad);

        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        // Measure calculation time (kernel execution)
        hipEventRecord(startCalc);
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
        hipEventRecord(stopCalc);
        hipEventSynchronize(stopCalc);

        float calcTime = 0;
        hipEventElapsedTime(&calcTime, startCalc, stopCalc);

        // Measure loading time for device to host
        hipEventRecord(startLoad);
        hipMemcpy(h_c.data(), d_c, N * sizeof(float), hipMemcpyDeviceToHost);
        hipEventRecord(stopLoad);
        hipEventSynchronize(stopLoad);

        float loadTimeBack = 0;
        hipEventElapsedTime(&loadTimeBack, startLoad, stopLoad);

        // Total loading time = H2D + D2H
        float totalLoadTime = loadTime + loadTimeBack;

        csv << N << "," << totalLoadTime << "," << calcTime << "\n";

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipEventDestroy(startLoad);
        hipEventDestroy(stopLoad);
        hipEventDestroy(startCalc);
        hipEventDestroy(stopCalc);
    }

    csv.close();
    std::cout << "Timing data saved to timings.csv\n";
    return 0;
}